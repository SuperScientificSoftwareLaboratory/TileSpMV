#include "hip/hip_runtime.h"
#include"common.h"
#include"mmio_highlevel.h"
//#include"mmio.h"
#include"utils.h"
#include"utils_tile.h"
#include"format_trans.h"
#include"spmv_tile.h"
#include"spmv_tile_balance.h"
#include"tilespmv_warp_bal.h"
//#include"tilespmv_warp_avx.h"
//#include"spmv_tile_balance_avx.h"
#include"LBLT.h"
#include"step.h"
#include"spmv_cuda.h"
#include <thrust/sort.h>


# define INDEX_DATA_TYPE unsigned char
//# define VAL_DATA_TYPE double

#define WARMUP_NUM 200

#define WARP_SIZE 32
#define WARP_PER_BLOCK 2

#define num_f 240
#define num_b 15

#define PREFETCH_SMEM_TH 8
#define COO_NNZ_TH 12

#define DEBUG_FORMATCOST 0

int main(int argc, char ** argv)
{

   
printf("--------------------------------!!-cuda-!!------------------------------------\n");
        Beidou_Tile_Matrix *matrixA_d = (Beidou_Tile_Matrix *)malloc(sizeof(Beidou_Tile_Matrix));
	//SMatrix matrixA_1;
    struct timeval t1, t2;
    int argi = 1;
 //   int BLOCK_SIZE;
 /*   if(argc > argi)
    {
        BLOCK_SIZE = atoi(argv[argi])  ;
        argi++;
    }

    printf(" -------------- BLOCK SIZE = %i --------------\n", BLOCK_SIZE);*/


    int nthreads;
    if(argc > argi)
    {
        nthreads = atoi(argv[argi]);
        argi++;
    }
omp_set_num_threads(nthreads);
    printf(" -------------- threads = %i --------------\n", nthreads);

    char  *filename;
    filename = argv[2];
    printf("MAT: -------------- %s --------------\n", filename);

    // load mtx A data to the csr format
    gettimeofday(&t1, NULL);
    mmio_allinone(&matrixA_d->m, &matrixA_d->n, &matrixA_d->nnz, &matrixA_d->isSymmetric, &matrixA_d->rowpointer, &matrixA_d->columnidx, &matrixA_d->value, filename);
    gettimeofday(&t2, NULL);
    double time_loadmat  = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("input matrix A: ( %i, %i ) nnz = %i\n loadfile time    = %4.5f sec\n", matrixA_d->m, matrixA_d->n, matrixA_d->nnz, time_loadmat/1000.0);

    for (int i = 0; i < matrixA_d->nnz; i++)
	    matrixA_d->value[i] = i % 10;


    matrixA_d->numtile =0;
    matrixA_d->tilem = matrixA_d->m%BLOCK_SIZE==0 ? matrixA_d->m/BLOCK_SIZE : (matrixA_d->m/BLOCK_SIZE)+1 ;
    matrixA_d->tilen = matrixA_d->n%BLOCK_SIZE==0 ? matrixA_d->n/BLOCK_SIZE : (matrixA_d->n/BLOCK_SIZE)+1 ;
 //MAT_PTR_TYPE *tile_ptr_A_1;    //block rowpointer of A
    matrixA_d->tile_ptr=(MAT_PTR_TYPE *)malloc((matrixA_d->tilem+1)*sizeof(MAT_PTR_TYPE));
    memset(matrixA_d->tile_ptr, 0, (matrixA_d->tilem+1)*sizeof(MAT_PTR_TYPE));

    int *new_coo_rowidx_1;
    int *new_coo_colidx_1;
    MAT_VAL_TYPE *new_coo_value_1;
   
    int *new_coocount_1;

    d_format_transform(matrixA_d, 
                &new_coo_value_1, &new_coo_colidx_1, &new_coo_rowidx_1, &new_coocount_1);
                
    Beidou_Tile_Matrix *matrixA = (Beidou_Tile_Matrix *)malloc(sizeof(Beidou_Tile_Matrix));
        mmio_allinone(&matrixA->m, &matrixA->n, &matrixA->nnz, &matrixA->isSymmetric, &matrixA->rowpointer, &matrixA->columnidx, &matrixA->value, filename);
            matrixA->numtile =0;
    matrixA->tilem = matrixA->m%BLOCK_SIZE==0 ? matrixA->m/BLOCK_SIZE : (matrixA->m/BLOCK_SIZE)+1 ;
    matrixA->tilen = matrixA->n%BLOCK_SIZE==0 ? matrixA->n/BLOCK_SIZE : (matrixA->n/BLOCK_SIZE)+1 ;
 //MAT_PTR_TYPE *tile_ptr_A_1;    //block rowpointer of A
    matrixA->tile_ptr=(MAT_PTR_TYPE *)malloc((matrixA->tilem+1)*sizeof(MAT_PTR_TYPE));
    memset(matrixA->tile_ptr, 0, (matrixA->tilem+1)*sizeof(MAT_PTR_TYPE));
/*int *new_coo_rowidx;
    int *new_coo_colidx;
    MAT_VAL_TYPE *new_coo_value;
    int *new_coocount;
    format_transform(matrixA, 
                  &new_coo_value, &new_coo_colidx, &new_coo_rowidx, &new_coocount);
    for(int i=0;i<matrixA_d->numtile;i++)
    {
        if(matrixA_d->tile_columnidx[i]!=matrixA->tile_columnidx[i]) 
        {
             printf("step2-error-colidx! i=%d   %d!=%d\n",i,matrixA->tile_columnidx[i],matrixA_d->tile_columnidx[i]);
            // break;
        }
        if(matrixA_d->tile_nnz[i]!=matrixA->tile_nnz[i]) 
        {
             printf("step2-error-nnz! i=%d   %d!=%d\n",i,matrixA->tile_nnz[i],matrixA_d->tile_nnz[i]);
             break;
        }
        for(int j=0;j<BLOCK_SIZE;j++)
        {
            if(matrixA->csr_ptr[i*BLOCK_SIZE+j]!=matrixA_d->csr_ptr_1[i*BLOCK_SIZE+j])
            {
               // printf("step2-error-ptr! i=%d   j=%d   %d!=%d\n",i,j,csr_ptr[i*BLOCK_SIZE+j],matrixA_d->csr_ptr_1[i*BLOCK_SIZE+j]);
                break;
            }
        }
        //printf("\n");
    }   */

 //balance
    int rowblkblock_1 = 0;
    int tilecnt_ave_1 =  (double) matrixA_d->tile_ptr[matrixA_d->tilem] / (double) matrixA_d->tilem;
    int *bal_num_gro = (int *) malloc(sizeof(int) * (matrixA_d->tilem +1));
    memset(bal_num_gro, 0, sizeof(int) * (matrixA_d->tilem +1)); 

  
    for (int blki = 0; blki < matrixA_d->tilem; blki++) 
    {
        int balancenumblk = matrixA_d->tile_ptr[blki + 1] - matrixA_d->tile_ptr[blki];
//        printf("balancenumblk = %i\n", balancenumblk); 
        if (balancenumblk <= tilecnt_ave_1)
        {
            bal_num_gro[blki]=rowblkblock_1;
            rowblkblock_1++;  
        }
        else 
        { 
            bal_num_gro[blki]=rowblkblock_1;
            rowblkblock_1 += ceil((double) balancenumblk / (double) tilecnt_ave_1);
        }
    }
    bal_num_gro[matrixA_d->tilem]=rowblkblock_1;
    printf("ave blk num = %4.2f, %i, %i\n", (double) matrixA_d->tile_ptr[matrixA_d->tilem] / (double) matrixA_d->tilem, matrixA_d->tilem, rowblkblock_1);


    matrixA_d->flag_bal_tile_rowidx = (unsigned int *) malloc(sizeof(unsigned int) * rowblkblock_1);
    memset(matrixA_d->flag_bal_tile_rowidx, 0, sizeof(unsigned int) * rowblkblock_1);
    unsigned int *d_flag_bal_tile_rowidx;
    hipMalloc((void **)&d_flag_bal_tile_rowidx, sizeof(unsigned int) * rowblkblock_1);
    hipMemcpy(d_flag_bal_tile_rowidx, matrixA_d->flag_bal_tile_rowidx, sizeof(unsigned int) * rowblkblock_1, hipMemcpyHostToDevice);


    matrixA_d->tile_bal_rowidx_colstart = (int *) malloc(sizeof(int) * rowblkblock_1);
    memset(matrixA_d->tile_bal_rowidx_colstart, 0, sizeof(int) * rowblkblock_1);
    int *d_tile_bal_rowidx_colstart;
    hipMalloc((void **)&d_tile_bal_rowidx_colstart, sizeof( int) * rowblkblock_1);
    hipMemcpy(d_tile_bal_rowidx_colstart, matrixA_d->tile_bal_rowidx_colstart, sizeof( int) * rowblkblock_1, hipMemcpyHostToDevice);

    matrixA_d->tile_bal_rowidx_colstop = (int *) malloc(sizeof(int) * rowblkblock_1);
    memset(matrixA_d->tile_bal_rowidx_colstop, 0, sizeof(int) * rowblkblock_1);
    int *d_tile_bal_rowidx_colstop;
    hipMalloc((void **)&d_tile_bal_rowidx_colstop, sizeof( int) * rowblkblock_1);
    hipMemcpy(d_tile_bal_rowidx_colstop, matrixA_d->tile_bal_rowidx_colstop, sizeof( int) * rowblkblock_1, hipMemcpyHostToDevice);

    int *group_ptr_1 = (int *) malloc(sizeof(int) * (rowblkblock_1 +1));
    memset(group_ptr_1, 0, sizeof(int) * (rowblkblock_1 +1));


    int *d_bal_num_gro;
    hipMalloc((void **)&d_bal_num_gro, sizeof(int) * (matrixA_d->tilem +1));
    hipMemcpy(d_bal_num_gro, bal_num_gro, sizeof(int) * (matrixA_d->tilem +1), hipMemcpyHostToDevice);

    MAT_PTR_TYPE *d_tile_ptr_A;
    hipMalloc((void **)&d_tile_ptr_A, sizeof(MAT_PTR_TYPE) *(matrixA_d->tilem+1) );
    hipMemcpy(d_tile_ptr_A, matrixA_d->tile_ptr, sizeof(MAT_PTR_TYPE) *(matrixA_d->tilem+1), hipMemcpyHostToDevice);

    int n_tile=32;
    double time_cuda_bal=0;
    int num_threads=0;
    int num_blocks=0;
    int num_tile_row=64;
    //int num_blocks=0;
    gettimeofday(&t1, NULL);
    
    for(int blki=0;blki<matrixA_d->tilem;blki+=n_tile)
    {   
        int start=blki;
        int end= blki+num_tile_row<matrixA_d->tilem ? end= blki+num_tile_row : end=matrixA_d->tilem;
         
        num_threads= end-start;
        num_blocks=num_threads/64+1;
   
        cuda_bal_step1<<< num_blocks, 64 >>>( matrixA_d->tilem, matrixA_d->tilen, d_tile_ptr_A,start,end,num_threads,
        d_flag_bal_tile_rowidx,d_tile_bal_rowidx_colstart,d_tile_bal_rowidx_colstop,d_bal_num_gro,tilecnt_ave_1);

        hipDeviceSynchronize();
        
     //   int length=i+n_tile>matrixA_d->tilem ? length=matrixA_d->tilem-i : length=n_tile ;
    }
    gettimeofday(&t2, NULL);
    time_cuda_bal = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
//printf("\n\n\n");
    printf("cuda-balance-step1 runtime    = %4.5f ms\n", time_cuda_bal);
  //   hipMemcpy(matrixA_d->tile_ptr+i, d_tile_ptr_A+i, sizeof(MAT_PTR_TYPE) *(length+1), hipMemcpyDeviceToHost);
    hipMemcpy(matrixA_d->tile_bal_rowidx_colstart, d_tile_bal_rowidx_colstart, sizeof( int) * rowblkblock_1, hipMemcpyDeviceToHost);
    hipMemcpy(matrixA_d->tile_bal_rowidx_colstop, d_tile_bal_rowidx_colstop, sizeof( int) * rowblkblock_1, hipMemcpyDeviceToHost);
    hipMemcpy(matrixA_d->flag_bal_tile_rowidx, d_flag_bal_tile_rowidx, sizeof(unsigned int) * rowblkblock_1, hipMemcpyDeviceToHost);


    for (int i = 0; i < rowblkblock_1; i ++)
    {
        int tile_start = matrixA_d->tile_bal_rowidx_colstart[i];
        int tile_stop = matrixA_d->tile_bal_rowidx_colstop[i];
        group_ptr_1[i] = matrixA_d->blknnz[tile_stop] - matrixA_d->blknnz[tile_start];
    }
    
    exclusive_scan(group_ptr_1,rowblkblock_1 +1);

    int *d_group_ptr;
    hipMalloc((void **)&d_group_ptr, sizeof( int) * rowblkblock_1);
    hipMemcpy(d_group_ptr, group_ptr_1, sizeof( int) * rowblkblock_1, hipMemcpyHostToDevice);

    int *flag_tilerow_start_1 = (int *)malloc((nthreads + 1) * sizeof(int));
    memset(flag_tilerow_start_1, 0, (nthreads) * sizeof(int));
    int *d_flag_tilerow_start;
    hipMalloc((void **)&d_flag_tilerow_start, (nthreads + 1) * sizeof(int));
    hipMemcpy(d_flag_tilerow_start, flag_tilerow_start_1, (nthreads + 1) * sizeof(int), hipMemcpyHostToDevice);

    int *flag_tilerow_stop_1 = (int *)malloc((nthreads) * sizeof(int));
    memset(flag_tilerow_stop_1, 0, (nthreads) * sizeof(int));

    //int *csrSplitter_normal = (int *)malloc((nthreads+1) * sizeof(int));
    int stridennz_1 = ceil((double)matrixA_d->nnz/(double)nthreads);

    
    double time_cuda_bal_1=0;
    gettimeofday(&t1, NULL);
    cuda_bal_step2<<< nthreads/64+1, 64 >>>( stridennz_1, matrixA_d->nnz,nthreads,rowblkblock_1,d_flag_tilerow_start,d_group_ptr);
    gettimeofday(&t2, NULL);
    hipDeviceSynchronize();
    time_cuda_bal_1 = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("cuda-balance-step2 runtime    = %4.5f ms\n", time_cuda_bal_1);
    hipMemcpy(flag_tilerow_start_1, d_flag_tilerow_start, (nthreads + 1) * sizeof(int), hipMemcpyDeviceToHost);
    for (int tid =0; tid < nthreads -1; tid ++)
    {
        flag_tilerow_stop_1[tid] = flag_tilerow_start_1[tid + 1];
    }
    flag_tilerow_stop_1[nthreads -1] = rowblkblock_1;




    for(int tid =0; tid < nthreads; tid ++)
    {
        printf("cuda-thread %i start = %i, stop = %i\n", tid, flag_tilerow_start_1[tid],flag_tilerow_stop_1[tid]);
       // printf("thread %i start = %i, stop = %i\n", tid, flag_tilerow_start_1[tid],flag_tilerow_stop_1[tid]);
    }    

    //extract coo data to a new matrix
    
    int nnz_1 = matrixA_d->coocount;
    printf("cuda-the number of coo data  = %i\n",nnz_1);

    double ratio_1 = (double)nnz_1/(double)matrixA_d->nnz;

    printf("cuda-the ratio of coo data  = %f\n",ratio_1);

    int *new_nnz_count_1 = (int *)malloc((matrixA_d->m+1) * sizeof(int));
    memset(new_nnz_count_1, 0,(matrixA_d->m+1) * sizeof(int));

    int *d_new_nnz_count;
    hipMalloc((void **)&d_new_nnz_count, (matrixA_d->m+1) * sizeof(int));
    hipMemcpy(d_new_nnz_count, new_nnz_count_1, (matrixA_d->m+1) * sizeof(int), hipMemcpyHostToDevice);

    int *d_new_coo_rowidx;
    hipMalloc((void **)&d_new_coo_rowidx, (matrixA_d->coocount) *sizeof(int));
    hipMemcpy(d_new_coo_rowidx, new_coo_rowidx_1, (matrixA_d->coocount) *sizeof(int), hipMemcpyHostToDevice);

    int num_nnz_1=4096;
    gettimeofday(&t1, NULL);

    for (int i=0;i<nnz_1;i+=num_nnz_1)//
    {
        //hipMemset(d_col_flag,0,tilenA*num_tile_row*16 * sizeof(unsigned char));
        int start=i;
        int end= i+num_nnz_1>nnz_1? end=nnz_1-1 : end=i+num_nnz_1-1;
        int num_threads= i+num_nnz_1>nnz_1? num_threads=nnz_1-i : num_threads=num_nnz_1;
        //int end= blki+num_tile_row<tilemA ? end= blki+num_tile_row : end=tilemA;
        num_blocks=(end-start)/32+1;
        //printf("end=%d  start=%d  num_blocks=%d  blki=%d\n",end,start,num_blocks,blki);
        cuda_coo_rowptrnum_kernel<<<num_blocks, 32 >>>(nnz_1,matrixA_d->m, matrixA_d->n,d_new_nnz_count,start,end,d_new_coo_rowidx,num_threads);
       hipDeviceSynchronize();
        
    }
    gettimeofday(&t2, NULL);
    hipDeviceSynchronize();
    double cuda_time_coo  = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("cuda_transform_coo_1 runtime    = %4.5f ms\n", cuda_time_coo);
    hipMemcpy(new_nnz_count_1, d_new_nnz_count, (matrixA_d->m+1) * sizeof(int), hipMemcpyDeviceToHost);

    int new_row_1 =0;

    //int *coo_new_rowidx_1 = (int *)malloc((matrixA_d->m ) * sizeof(int));
    matrixA_d->coo_new_rowidx = (int *)malloc((matrixA_d->m ) * sizeof(int));
    memset(matrixA_d->coo_new_rowidx, 0, matrixA_d->m * sizeof(int));

    for (int i=0; i < matrixA_d->m ; i ++)
    {
        if (new_nnz_count_1[i] !=0){
            matrixA_d->coo_new_rowidx[new_row_1++] = i;
        }
    }

    //int *coo_new_matrix_ptr_1 = (int*)malloc(sizeof(int)*(new_row_1+1));
    matrixA_d->coo_new_matrix_ptr = (int*)malloc(sizeof(int)*(new_row_1+1));
    memset(matrixA_d->coo_new_matrix_ptr, 0, (new_row_1 + 1) * sizeof(int));
    int cnt_1 =0;

    for (int i=0; i < matrixA_d->m ; i ++)
    {
        if (new_nnz_count_1[i] !=0){
            matrixA_d->coo_new_matrix_ptr[cnt_1++] = new_nnz_count_1[i] ;
        }
    }
    exclusive_scan(matrixA_d->coo_new_matrix_ptr,new_row_1+1);
    exclusive_scan(new_nnz_count_1, matrixA_d->m +1);
   // exclusive_scan(new_nnz_count_1, rowA +1);

    matrixA_d->coo_new_matrix_colidx  = (int *)malloc(nnz_1 * sizeof(int));
    memset(matrixA_d->coo_new_matrix_colidx, 0,  nnz_1 * sizeof(int));
    int *d_coo_new_colidx;
    hipMalloc((void **)&d_coo_new_colidx, nnz_1 * sizeof(int));
    hipMemcpy(d_coo_new_colidx, matrixA_d->coo_new_matrix_colidx, nnz_1 * sizeof(int), hipMemcpyHostToDevice);
//coo_new_matrix_ptr
    //MAT_VAL_TYPE *coo_new_value_1 = (MAT_VAL_TYPE *)malloc(nnz_1 * sizeof(MAT_VAL_TYPE));
    matrixA_d->coo_new_matrix_value = (MAT_VAL_TYPE *)malloc(nnz_1 * sizeof(MAT_VAL_TYPE));
    memset(matrixA_d->coo_new_matrix_value, 0,  nnz_1 * sizeof(MAT_VAL_TYPE));
    MAT_VAL_TYPE *d_coo_new_value;
    hipMalloc((void **)&d_coo_new_value, nnz_1 * sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_coo_new_value, matrixA_d->coo_new_matrix_value, nnz_1 * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

    int *new_num_1  = (int *)malloc( matrixA_d->m* sizeof(int));//new_row_1
    memset(new_num_1, 0,  matrixA_d->m * sizeof(int));
    int *d_new_num;
    hipMalloc((void **)&d_new_num, matrixA_d->m* sizeof(int));
    hipMemcpy(d_new_num, new_num_1, matrixA_d->m* sizeof(int), hipMemcpyHostToDevice);
   // hipMemcpy(d_new_coo_colidx, new_coo_colidx_1, (matrixA_d->coocount) *sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(d_new_coo_value, new_coo_value_1, (matrixA_d->coocount) *sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
  
    int *d_new_coo_colidx;
    hipMalloc((void **)&d_new_coo_colidx, (nnz_1) *sizeof(int));
    hipMemcpy(d_new_coo_colidx, new_coo_colidx_1, (nnz_1) *sizeof(int), hipMemcpyHostToDevice);

    MAT_VAL_TYPE *d_new_coo_value;
    hipMalloc((void **)&d_new_coo_value, (nnz_1) *sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_new_coo_value, new_coo_value_1, (nnz_1) *sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
    //hipMemcpy(d_new_coo_value, new_coo_value_1, (matrixA_d->coocount) *sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_new_nnz_count, new_nnz_count_1, (matrixA_d->m+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_new_coo_rowidx, new_coo_rowidx_1, (matrixA_d->coocount) *sizeof(int), hipMemcpyHostToDevice);
    gettimeofday(&t1, NULL);
   // int x=0;
    for (int i=0;i<nnz_1;i+=num_nnz_1)//
    {
        //hipMemset(d_col_flag,0,tilenA*num_tile_row*16 * sizeof(unsigned char));
        int start=i;
        int end= i+num_nnz_1>nnz_1? end=nnz_1-1 : end=i+num_nnz_1-1;
        int num_threads= i+num_nnz_1>nnz_1? num_threads=nnz_1-i : num_threads=num_nnz_1;
        //int end= blki+num_tile_row<tilemA ? end= blki+num_tile_row : end=tilemA;
        num_blocks=(end-start)/32+1;
       // printf("end=%d  start=%d \n",end,start);
        cuda_coo_kernel<<<num_blocks, 32 >>>(nnz_1,matrixA_d->m, matrixA_d->n,d_new_nnz_count,start,end,d_new_coo_rowidx,num_threads,
                 d_coo_new_colidx,d_coo_new_value,d_new_num,d_new_coo_value,d_new_coo_colidx);
       hipDeviceSynchronize();
        
    }
    gettimeofday(&t2, NULL);
    hipDeviceSynchronize();
    double cuda_time_coo_1  = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("cuda_transform_coo_2 runtime    = %4.5f ms\n", cuda_time_coo_1);
    hipMemcpy(matrixA_d->coo_new_matrix_colidx, d_coo_new_colidx, nnz_1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(matrixA_d->coo_new_matrix_value, d_coo_new_value, nnz_1 * sizeof(MAT_VAL_TYPE), hipMemcpyDeviceToHost);
    hipMemcpy(new_num_1, d_new_num, matrixA_d->m* sizeof(int), hipMemcpyDeviceToHost);
//printf("new_coo_rowidx[nnz]=%d\n",new_coo_rowidx[matrixA_d->coocount-1]);
 //printf("hhhh\n");
//sort
    gettimeofday(&t1, NULL);
 //  #pragma omp parallel for
   // for (int i =0 ; i < new_row_1; i ++)
    //{
      //  int nnz_offset = coo_new_matrix_ptr_1[i];
        //int length = coo_new_matrix_ptr_1[i+1] - coo_new_matrix_ptr_1[i];

      //  quick_sort_key_val_pair(coo_new_colidx_1 + nnz_offset, coo_new_value_1 + nnz_offset, length);
   // }
    //thrust::sort_by_key(new_coo_colidx_1 , new_coo_colidx_1 +nnz_1, new_coo_value_1);
   // sort_by_row_and_column(new_coo_rowidx_1,new_coo_colidx_1,new_coo_value_1,0,nnz_1,0,nnz_1);
    for (int i =0 ; i < new_row_1; i ++)
    {
        int nnz_offset = matrixA_d->coo_new_matrix_ptr[i];
        int length = matrixA_d->coo_new_matrix_ptr[i+1] - matrixA_d->coo_new_matrix_ptr[i];

        //quick_sort_key_val_pair(matrixA_d->coo_new_matrix_colidx + nnz_offset, matrixA_d->coo_new_matrix_value + nnz_offset, length);
        thrust::sort_by_key(matrixA_d->coo_new_matrix_colidx + nnz_offset, matrixA_d->coo_new_matrix_colidx + nnz_offset + length, matrixA_d->coo_new_matrix_value + nnz_offset);
    }

    gettimeofday(&t2, NULL);
    double time_sort_1  = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("cuda-sort use   %4.5f ms\n", time_sort_1);


printf("--------------------------------!!-cuda-end-!!------------------------------------\n");

    MAT_VAL_TYPE *x = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * matrixA_d->n);
	for (int i = 0; i < matrixA_d->n; i++)
	{
		x[i] = i % 10;
	}
MAT_VAL_TYPE *y = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * matrixA_d->m);
    memset(y, 0, sizeof(MAT_VAL_TYPE) * matrixA_d->m);

    gettimeofday(&t1, NULL);
    for (int repeat =0; repeat < BENCH_REPEAT; repeat ++)
    {
            memset(y, 0, sizeof(MAT_VAL_TYPE) * matrixA_d->m);

        tilespmv(matrixA_d,  x, y,new_row_1);

    }
    gettimeofday(&t2, NULL);
    double time_tile  = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("spmv runtime using tile    = %4.5f ms\n", time_tile/BENCH_REPEAT);
    
    MAT_VAL_TYPE *y_golden = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * matrixA_d->m);
    memset(y_golden, 0, sizeof(MAT_VAL_TYPE) * matrixA_d->m);
    for (int i = 0; i < matrixA_d->n; i++)
	{
		MAT_VAL_TYPE sum = 0;
		for (int j = matrixA_d->rowpointer[i]; j < matrixA_d->rowpointer[i+1]; j++)
		{
			sum += matrixA_d->value[j] * x[matrixA_d->columnidx[j]];
		}
		y_golden[i] = sum;
	}

printf("--------------------------------!!-bal-!!------------------------------------\n");
    //int *csr_ptr_1 = (int *)malloc(((matrixA_d->numtile + 1) * BLOCK_SIZE) * sizeof(int));
    matrixA_d->csr_ptr = (unsigned char *)malloc(((matrixA_d->numtile+1) * BLOCK_SIZE) * sizeof(unsigned char));
    for(int i=0;i<matrixA_d->numtile;i++)
    {
        matrixA_d->csr_ptr[i]=matrixA_d->csr_ptr_1[i];
    }
	double *Ysum_1 = (double *)malloc(sizeof(double) * nthreads);
	memset (Ysum_1, 0, sizeof(double) * nthreads);
    double *Ypartialsum_1 = (double *)malloc(sizeof(double) * nthreads);
	memset (Ypartialsum_1, 0, sizeof(double) * nthreads);
    MAT_VAL_TYPE *y_bal_1 = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * matrixA_d->m);
    memset(y_bal_1, 0, sizeof(MAT_VAL_TYPE) * matrixA_d->m);

    matrixA_d->csrSplitter_yid = (int *)malloc((nthreads+1) * sizeof(int));
    matrixA_d->Yid = (int *)malloc(sizeof(int) * nthreads);
    matrixA_d->Start1 = (int *)malloc(sizeof(int) * nthreads);
	matrixA_d->End1 = (int *)malloc(sizeof(int) * nthreads);
	matrixA_d->label = (int *)malloc(sizeof(int) * nthreads);
    matrixA_d->Start2 = (int *)malloc(sizeof(int) * nthreads);
	matrixA_d->End2 = (int *)malloc(sizeof(int) * nthreads);
    /*tilespmv_balance(matrixA_d, rowblkblock_1,  csr_ptr_1, hyb_coocount_1, nnz_temp, tile_count_temp,csr_offset_1, csrptr_offset_1, 
    coo_offset_1, ell_offset_1, hyb_offset_1, dns_offset_1, dnsrow_offset_1, dnscol_offset_1, x, y_bal_1,
    flag_tilerow_start_1, flag_tilerow_stop_1);*/
    spmvLBLT(new_row_1,nthreads, matrixA_d->m, matrixA_d->n, matrixA_d->nnz, matrixA_d->coo_new_rowidx,matrixA_d->coo_new_matrix_ptr,
            matrixA_d->coo_new_matrix_colidx,matrixA_d->coo_new_matrix_value,matrixA_d->csrSplitter_yid,matrixA_d->Yid,
            matrixA_d->Start1,matrixA_d->End1,matrixA_d->label,matrixA_d->Start2,matrixA_d->End2);
    //printf("hhhhhhhhhhh\n");
    tilespmv_balance(matrixA_d, rowblkblock_1, x, y_bal_1,
                        flag_tilerow_start_1, flag_tilerow_stop_1, Ysum_1, Ypartialsum_1);
    int balcnt_1 =0;
//printf("hhhhhhhhhhh\n");
    for (int i=0; i < matrixA_d->m; i ++)//
    {
        if (y_golden[i] != y_bal_1[i])
        {
          // printf("y[%i] = %f, y_bal_1[%i] = %f\n", i, y[i], i, y_bal_1[i]);
            balcnt_1 ++;
        }
    }

    printf("cuda-balance result errcnt = %i\n",balcnt_1);
    //printf("omp_get_max_threads()=%d\n",omp_get_max_threads());
printf("--------------------------------!!-bal end-!!------------------------------------\n");
printf("--------------------------------!!-cuda spmv-!!------------------------------------\n");
	char *flag=(char *)malloc(matrixA_d->tilen*sizeof(char));
	int nnzbl=0;

	for (int i=0;i<matrixA_d->tilem;i++)
	{
		memset(flag,0,matrixA_d->tilen*sizeof(char));
		int start= i*BLOCK_SIZE;
		int end = i==matrixA_d->tilem-1 ?  matrixA_d->m : (i+1)*BLOCK_SIZE ;
		for (int j=matrixA_d->rowpointer[start];j<matrixA_d->rowpointer[end];j++)
		{
			int jc=matrixA_d->columnidx[j]/BLOCK_SIZE;
			if (flag[jc]==0)
			{
				flag[jc]=1;
				nnzbl++;
			}
		} 
	}
	int colid=0;
    int ptrA_length=0;
    for (int i=0;i<matrixA_d->tilem;i++)
	{
        memset(flag,0,matrixA_d->tilen*sizeof(char));
        int start= i*BLOCK_SIZE;
        int end = i==matrixA_d->tilem-1 ?  matrixA_d->m : (i+1)*BLOCK_SIZE ;
		for (int j=matrixA_d->rowpointer[start];j<matrixA_d->rowpointer[end];j++)
        {
            int jc=matrixA_d->columnidx[j]/BLOCK_SIZE;
            if (flag[jc]==0)
            {
                flag[jc]=1;
              //  rowblock_ptr[i+1]++;
             //   columnid[colid]=jc;
                colid++;
                ptrA_length+=(end-start);
            }
	    } 
	}
    char *d_flag; 
	hipMalloc((void **)&d_flag, matrixA_d->tilen*sizeof(char));
	hipMemcpy(d_flag, flag, matrixA_d->tilen*sizeof(char), hipMemcpyHostToDevice);

    MAT_VAL_TYPE *y_d = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * matrixA_d->m);
    memset(y_d, 0, sizeof(MAT_VAL_TYPE) * matrixA_d->m);

    MAT_VAL_TYPE *d_x;
    MAT_VAL_TYPE *d_y;

    hipMalloc((void **)&d_x, matrixA_d->n * sizeof(MAT_VAL_TYPE)); 
    hipMalloc((void **)&d_y, matrixA_d->m * sizeof(MAT_VAL_TYPE));

    hipMemcpy(d_x, x, matrixA_d->n * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y_d, matrixA_d->m * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);



  //  MAT_PTR_TYPE *d_tile_ptr_A;
  //  hipMalloc((void **)&d_tile_ptr_A, sizeof(MAT_PTR_TYPE) *(matrixA_d->tilem+1) );
    hipMemcpy(d_tile_ptr_A, matrixA_d->tile_ptr, sizeof(MAT_PTR_TYPE) *(matrixA_d->tilem+1), hipMemcpyHostToDevice);

    int *d_tile_columnidx;
    hipMalloc((void **)&d_tile_columnidx, (matrixA_d->numtile + 1) * sizeof(int) );
    hipMemcpy(d_tile_columnidx, matrixA_d->tile_columnidx, (matrixA_d->numtile+1) * sizeof(int), hipMemcpyHostToDevice);

    int *d_tile_nnz;
    hipMalloc((void **)&d_tile_nnz, (matrixA_d->numtile + 1) * sizeof(int) );
    hipMemcpy(d_tile_nnz, matrixA_d->tile_nnz, (matrixA_d->numtile+1) * sizeof(int), hipMemcpyHostToDevice);

    char *d_Format;
    hipMalloc((void **)&d_Format, matrixA_d->numtile* sizeof(char) );
    hipMemcpy(d_Format, matrixA_d->Format, matrixA_d->numtile* sizeof(char), hipMemcpyHostToDevice);

    int *d_blknnz;
    hipMalloc((void **)&d_blknnz, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_blknnz, matrixA_d->blknnz, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

    int *d_dns_offset;
    hipMalloc((void **)&d_dns_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_dns_offset, matrixA_d->dns_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);
printf("matrixA_d->dns_offset=%d  %d\n",matrixA_d->dns_offset[matrixA_d->numtile],matrixA_d->dns_offset[matrixA_d->numtile-1]);


    int *d_denserowptr;
    hipMalloc((void **)&d_denserowptr, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_denserowptr, matrixA_d->denserowptr, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);
    
    int *d_dnsrow_offset;
    hipMalloc((void **)&d_dnsrow_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_dnsrow_offset, matrixA_d->dnsrow_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);
    
    int *d_densecolptr;
    hipMalloc((void **)&d_densecolptr, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_densecolptr, matrixA_d->densecolptr, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);
    
    int *d_dnscol_offset;
    hipMalloc((void **)&d_dnscol_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_dnscol_offset, matrixA_d->dnscol_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

    int *d_csr_offset;
    hipMalloc((void **)&d_csr_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_csr_offset, matrixA_d->csr_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

    int *d_csrptr_offset;
    hipMalloc((void **)&d_csrptr_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_csrptr_offset, matrixA_d->csrptr_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

    int *d_ell_offset;
    hipMalloc((void **)&d_ell_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_ell_offset, matrixA_d->ell_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

    int *d_coo_offset;
    hipMalloc((void **)&d_coo_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_coo_offset, matrixA_d->coo_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

    char *d_blkwidth;
    hipMalloc((void **)&d_blkwidth, (matrixA_d->numtile + 1)* sizeof(char) );
    hipMemcpy(d_blkwidth, matrixA_d->blkwidth, (matrixA_d->numtile + 1)* sizeof(char), hipMemcpyHostToDevice);

    int *d_hyb_coocount;
    hipMalloc((void **)&d_hyb_coocount, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_hyb_coocount, matrixA_d->hyb_coocount, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

    int *d_hyb_offset;
    hipMalloc((void **)&d_hyb_offset, (matrixA_d->numtile + 1)* sizeof(int) );
    hipMemcpy(d_hyb_offset, matrixA_d->hyb_offset, (matrixA_d->numtile + 1)* sizeof(int), hipMemcpyHostToDevice);

//CSR
    MAT_VAL_TYPE *d_Tile_csr_Val;
    hipMalloc((void **)&d_Tile_csr_Val, (matrixA_d->csrsize)*sizeof(MAT_VAL_TYPE) );
    hipMemcpy(d_Tile_csr_Val, matrixA_d->Tile_csr_Val, (matrixA_d->csrsize)*sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

    unsigned char  *d_Tile_csr_Col;
    hipMalloc((void **)&d_Tile_csr_Col,(matrixA_d->csrsize)*sizeof(unsigned char) );
    hipMemcpy(d_Tile_csr_Col, matrixA_d->Tile_csr_Col, (matrixA_d->csrsize)*sizeof(unsigned char), hipMemcpyHostToDevice);

    unsigned char *d_Tile_csr_Ptr;
    hipMalloc((void **)&d_Tile_csr_Ptr,(matrixA_d->csrtilecount * BLOCK_SIZE)*sizeof(unsigned char) );
    hipMemcpy(d_Tile_csr_Ptr, matrixA_d->Tile_csr_Ptr,(matrixA_d->csrtilecount * BLOCK_SIZE)*sizeof(unsigned char), hipMemcpyHostToDevice);

//ELL
    MAT_VAL_TYPE *d_Tile_ell_Val;
    hipMalloc((void **)&d_Tile_ell_Val, (matrixA_d->ellsize)*sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_Tile_ell_Val, matrixA_d->Tile_ell_Val, (matrixA_d->ellsize)*sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

    unsigned char *d_Tile_ell_colIdx;
    hipMalloc((void **)&d_Tile_ell_colIdx, (matrixA_d->ellsize)*sizeof(unsigned char));
    hipMemcpy(d_Tile_ell_colIdx, matrixA_d->Tile_ell_colIdx, (matrixA_d->ellsize)*sizeof(unsigned char), hipMemcpyHostToDevice);

//HYB
    MAT_VAL_TYPE *d_Tile_hyb_Val;
    hipMalloc((void **)&d_Tile_hyb_Val, (matrixA_d->hybellsize+matrixA_d->hybcoosize)*sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_Tile_hyb_Val, matrixA_d->Tile_hyb_Val, (matrixA_d->hybellsize+matrixA_d->hybcoosize)*sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

    unsigned char *d_Tile_hyb_ellcolIdx;
    hipMalloc((void **)&d_Tile_hyb_ellcolIdx, (matrixA_d->hybellsize+matrixA_d->hybcoosize)*sizeof(unsigned char));
    hipMemcpy(d_Tile_hyb_ellcolIdx, matrixA_d->Tile_hyb_ellcolIdx, (matrixA_d->hybellsize+matrixA_d->hybcoosize)*sizeof(unsigned char), hipMemcpyHostToDevice);


    unsigned char *d_Tile_hyb_coorowIdx;
    hipMalloc((void **)&d_Tile_hyb_coorowIdx, (matrixA_d->hybcoosize)*sizeof(unsigned char));
    hipMemcpy(d_Tile_hyb_coorowIdx, matrixA_d->Tile_hyb_coorowIdx, (matrixA_d->hybcoosize)*sizeof(unsigned char), hipMemcpyHostToDevice);

//dense
    MAT_VAL_TYPE *d_Tile_dns_Val;
    hipMalloc((void **)&d_Tile_dns_Val, (matrixA_d->dense_size)*sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_Tile_dns_Val, matrixA_d->Tile_dns_Val, (matrixA_d->dense_size)*sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

//dense row
    MAT_VAL_TYPE *d_Tile_dnsrow_Val;
    hipMalloc((void **)&d_Tile_dnsrow_Val, (matrixA_d->denserow_size) * sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_Tile_dnsrow_Val, matrixA_d->Tile_dnsrow_Val, (matrixA_d->denserow_size) * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

    char *d_Tile_dnsrow_idx ;
    hipMalloc((void **)&d_Tile_dnsrow_idx, matrixA_d->denserowptr[matrixA_d->numtile] * sizeof(char));
    hipMemcpy(d_Tile_dnsrow_idx, matrixA_d->Tile_dnsrow_idx, matrixA_d->denserowptr[matrixA_d->numtile] * sizeof(char), hipMemcpyHostToDevice);

//dense col
    MAT_VAL_TYPE *d_Tile_dnscol_Val;
    hipMalloc((void **)&d_Tile_dnscol_Val, (matrixA_d->densecol_size) * sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_Tile_dnscol_Val, matrixA_d->Tile_dnscol_Val, (matrixA_d->densecol_size) * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

    char *d_Tile_dnscol_idx;
    hipMalloc((void **)&d_Tile_dnscol_idx, matrixA_d->densecolptr[matrixA_d->numtile] * sizeof(char));
    hipMemcpy(d_Tile_dnscol_idx, matrixA_d->Tile_dnscol_idx, matrixA_d->densecolptr[matrixA_d->numtile] * sizeof(char), hipMemcpyHostToDevice);

//COO
 //   MAT_VAL_TYPE *d_coo_new_value;
 //   hipMalloc((void **)&d_coo_new_value, nnz_1 * sizeof(MAT_VAL_TYPE));
    hipMemcpy(d_coo_new_value, matrixA_d->coo_new_matrix_value, nnz_1 * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);

    int *d_coo_new_rowidx;
    hipMalloc((void **)&d_coo_new_rowidx, (matrixA_d->m ) * sizeof(int));
    hipMemcpy(d_coo_new_rowidx, matrixA_d->coo_new_rowidx, (matrixA_d->m ) * sizeof(int), hipMemcpyHostToDevice);

    int *d_coo_new_matrix_ptr;
    hipMalloc((void **)&d_coo_new_matrix_ptr, sizeof(int)*(new_row_1+1));
    hipMemcpy(d_coo_new_matrix_ptr, matrixA_d->coo_new_matrix_ptr, sizeof(int)*(new_row_1+1), hipMemcpyHostToDevice);

    int *d_coo_new_matrix_colidx;
    hipMalloc((void **)&d_coo_new_matrix_colidx, nnz_1 * sizeof(int));
    hipMemcpy(d_coo_new_matrix_colidx, matrixA_d->coo_new_matrix_colidx, nnz_1 * sizeof(int), hipMemcpyHostToDevice);

    unsigned char *d_blknnznnz;
    hipMalloc((void **)&d_blknnznnz, (matrixA_d->numtile + 1)* sizeof(unsigned char));
    hipMemcpy(d_blknnznnz, matrixA_d->blknnznnz, (matrixA_d->numtile + 1)* sizeof(unsigned char), hipMemcpyHostToDevice);

    int *d_coodeferoffset;
    int *d_deferbuf_coooff;
    int *d_deferbuf_dxoff;

    hipMalloc((void **)&d_coodeferoffset, rowblkblock_1 * sizeof(int));
    hipMemset(d_coodeferoffset, 0, rowblkblock_1 * sizeof(int));

    hipMalloc((void **)&d_deferbuf_coooff, rowblkblock_1 * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));
    hipMemset(d_deferbuf_coooff, 0, rowblkblock_1 * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));
    hipMalloc((void **)&d_deferbuf_dxoff, rowblkblock_1 * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));
    hipMemset(d_deferbuf_dxoff, 0, rowblkblock_1 * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));

    //int *d_flag_tilerow_start;
    //hipMalloc((void **)&d_flag_tilerow_start, (nthreads + 1) * sizeof(int));
    hipMemcpy(d_flag_tilerow_start, flag_tilerow_start_1, (rowblkblock_1 + 1) * sizeof(int), hipMemcpyHostToDevice);
    int *d_flag_tilerow_stop;
    hipMalloc((void **)&d_flag_tilerow_stop, (rowblkblock_1 ) * sizeof(int));
    hipMemcpy(d_flag_tilerow_stop, flag_tilerow_stop_1, (rowblkblock_1 ) * sizeof(int), hipMemcpyHostToDevice);

//-------------------
    // analysis
    int rowblkblock = 0;
    //int iiiii = 0;
    for (int blki = 0; blki < matrixA_d->tilem; blki++)
    {
        int balancenumblk = matrixA_d->tile_ptr[blki+1] - matrixA_d->tile_ptr[blki];
        if (balancenumblk <= PREFETCH_SMEM_TH) 
            rowblkblock++;
        else 
        {
            rowblkblock += ceil((double)balancenumblk / (double)PREFETCH_SMEM_TH);
            //printf("[%i] blki = %i, balancenumblk = %i, rowblkblock += %i\n", iiiii, blki, balancenumblk, balancenumblk / 32); 
            //iiiii++;
        }

    }
    printf("ave blk num = %4.2f, %i, %i\n", (double)matrixA_d->tile_ptr[matrixA_d->tilem] / (double)matrixA_d->tilem, matrixA_d->tilem, rowblkblock);

    unsigned int * blkcoostylerowidx = (unsigned int *)malloc(sizeof(unsigned int) * rowblkblock);
    memset(blkcoostylerowidx, 0, sizeof(unsigned int) * rowblkblock);
    int * blkcoostylerowidx_colstart = (int *)malloc(sizeof(int) * rowblkblock);
    memset(blkcoostylerowidx_colstart, 0, sizeof(int) * rowblkblock);
    int * blkcoostylerowidx_colstop = (int *)malloc(sizeof(int) * rowblkblock);
    memset(blkcoostylerowidx_colstop, 0, sizeof(int) * rowblkblock);

    int rowblkblockcnt = 0;
    for (int blki = 0; blki < matrixA_d->tilem; blki++)
    {
        int balancenumblk = matrixA_d->tile_ptr[blki+1] - matrixA_d->tile_ptr[blki];
//printf("blki=%d  balancenumblk=%d\n",blki,balancenumblk);
        if (balancenumblk <= PREFETCH_SMEM_TH) 
        {
            blkcoostylerowidx[rowblkblockcnt] = blki;
            rowblkblockcnt++;
        }
        else 
        {
            int numblklocal = ceil((double)balancenumblk / (double)PREFETCH_SMEM_TH);
            int lenblklocal = ceil((double)balancenumblk / (double)numblklocal);
            for (int iii = 0; iii < numblklocal; iii++)
            {
                blkcoostylerowidx[rowblkblockcnt] = blki | 0x80000000; // can generate -0
                blkcoostylerowidx_colstart[rowblkblockcnt] = matrixA_d->tile_ptr[blki] + iii * lenblklocal;
                if (iii == numblklocal - 1)
                    blkcoostylerowidx_colstop[rowblkblockcnt] = matrixA_d->tile_ptr[blki] + balancenumblk;
                else 
                    blkcoostylerowidx_colstop[rowblkblockcnt] = matrixA_d->tile_ptr[blki] + (iii+1) * lenblklocal;

                rowblkblockcnt++;
            }
        }

    }



    unsigned int * d_blkcoostylerowidx;
    int * d_blkcoostylerowidx_colstart;
    int * d_blkcoostylerowidx_colstop;

    hipMalloc((void **)&d_blkcoostylerowidx, rowblkblock * sizeof(unsigned int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstart, rowblkblock * sizeof(int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstop, rowblkblock * sizeof(int));

    hipMemcpy(d_blkcoostylerowidx, blkcoostylerowidx, rowblkblock * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstart, blkcoostylerowidx_colstart, rowblkblock * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstop, blkcoostylerowidx_colstop, rowblkblock * sizeof(int), hipMemcpyHostToDevice);
//-------------------

    //num_threads = WARP_PER_BLOCK * WARP_SIZE;
    //num_blocks = ceil ((double)rowblkblock_1 / (double)(num_threads / WARP_SIZE));

    double time_cuda_spmv6 = 0;
    for (int i = 0; i < 1; i++)//
    {
        int num_threads = WARP_PER_BLOCK * WARP_SIZE;
        int num_blocks = ceil ((double)rowblkblock / (double)(num_threads / WARP_SIZE));
      //  printf("num_threads=%d  num_blocks=%d rowblkblock_1=%d  rowblkblock=%d\n",num_threads,num_blocks,rowblkblock_1,rowblkblock);
        hipMemset(d_y, 0, matrixA_d->m * sizeof(MAT_VAL_TYPE));
       // rowblkblock_1=10057;
        gettimeofday(&t1, NULL);
        stir_spmv_cuda_kernel_v6<<< num_blocks, num_threads >>>
                (matrixA_d->tilem,matrixA_d->tilen,matrixA_d->m,matrixA_d->n,matrixA_d->dense_size,
                d_tile_nnz,  d_flag,  d_tile_ptr_A,  d_tile_columnidx,  d_Format,  d_blknnz, d_blknnznnz,
                d_Tile_csr_Col, d_Tile_csr_Val,  d_Tile_csr_Ptr, 
                d_blkwidth, d_Tile_ell_Val, d_Tile_ell_colIdx, 
                d_Tile_hyb_ellcolIdx, d_Tile_hyb_Val, 
                d_Tile_dns_Val, 
                d_denserowptr,  d_Tile_dnsrow_Val,  d_Tile_dnsrow_idx, 
                d_densecolptr,  d_Tile_dnscol_Val,  d_Tile_dnscol_idx, 
                d_dns_offset, d_dnsrow_offset, d_dnscol_offset, d_csr_offset, d_csrptr_offset, d_ell_offset, d_coo_offset, d_hyb_coocount, d_hyb_offset,
                rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
//d_flag_bal_tile_rowidx, d_flag_tilerow_start, d_flag_tilerow_stop,
                d_coo_new_rowidx, d_coo_new_matrix_colidx, d_coo_new_value,d_coo_new_matrix_ptr,
                d_x,  d_y, 7, d_coodeferoffset, d_deferbuf_coooff, d_deferbuf_dxoff);
        hipDeviceSynchronize();
        gettimeofday(&t2, NULL);

        time_cuda_spmv6 += (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
        num_threads = WARP_PER_BLOCK * WARP_SIZE;
        num_blocks = ceil ((double)new_row_1 / (double)num_threads);

    hipMemcpy(y_d, d_y, matrixA_d->m * sizeof(MAT_VAL_TYPE), hipMemcpyDeviceToHost);

  //  hipMemcpy(d_y, y_d, matrixA_d->m * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
hipMemcpy(d_y, y_d, matrixA_d->m * sizeof(MAT_VAL_TYPE), hipMemcpyHostToDevice);
gettimeofday(&t1, NULL);
        spmv_coo<<< num_blocks, num_threads >>>(d_coo_new_matrix_ptr, d_coo_new_matrix_colidx, d_coo_new_value,d_coo_new_rowidx,matrixA_d->m,new_row_1,d_x,  d_y);

       hipDeviceSynchronize();
        gettimeofday(&t2, NULL);
time_cuda_spmv6 += (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;


//printf("num_threads=%d  num_blocks=%d rowblkblock_1=%d  rowblkblock=%d\n",num_threads,num_blocks,rowblkblock_1,rowblkblock);
    }

  //  time_cuda_spmv6 /= BENCH_REPEAT;
    printf("\n  CUDA SpMV V6 %f ms, %f GFlops\n", 
        time_cuda_spmv6, 2 * (double)matrixA_d->nnz * 1.0e-6 / time_cuda_spmv6);


hipMemcpy(y_d, d_y, matrixA_d->m * sizeof(MAT_VAL_TYPE), hipMemcpyDeviceToHost);




    int cudcnt=0;
    for (int i=0; i < matrixA_d->m; i ++)//

    {
        if (y_golden[i] != y_d[i])
        {
           //printf("y[%i] = %f, y_d[%i] = %f    %f\n", i, y[i], i, y_d[i],y[i]-y_d[i]);
            cudcnt ++;
        }
    }

    printf("cuda-balance result errcnt = %i\n",cudcnt);


    FILE *fouttime = fopen("beidouspmv_res.csv", "a");
	fprintf(fouttime, "%s,%i,%i,%i,%i,%f,%f,%i\n",
			filename, matrixA_d->m, matrixA_d->n, matrixA_d->nnz, matrixA_d->numtile,
			time_cuda_spmv6, 2 * (double)matrixA_d->nnz * 1.0e-6 / time_cuda_spmv6, cudcnt);
fclose(fouttime);
    hipFree(d_tile_ptr_A);
    hipFree(d_flag);
    hipFree(d_tile_columnidx);
    hipFree(d_tile_nnz);
    hipFree(d_Format);
    hipFree(d_blknnz);
    hipFree(d_blknnznnz);

    hipFree(d_dns_offset);
    hipFree(d_denserowptr);
    hipFree(d_dnsrow_offset);
    hipFree(d_densecolptr);
    hipFree(d_dnscol_offset);
    hipFree(d_csr_offset);
    hipFree(d_csrptr_offset);
    hipFree(d_ell_offset);
    hipFree(d_coo_offset);
    hipFree(d_blkwidth);
    hipFree(d_hyb_coocount);
    hipFree(d_hyb_offset);

    hipFree(d_Tile_csr_Val);
    hipFree(d_Tile_csr_Col);
    hipFree(d_Tile_csr_Ptr);
    hipFree(d_Tile_ell_Val);
    hipFree(d_Tile_ell_colIdx);
    hipFree(d_Tile_hyb_Val);
    hipFree(d_Tile_hyb_ellcolIdx);
    hipFree(d_Tile_hyb_coorowIdx);
    hipFree(d_Tile_dns_Val);
    hipFree(d_Tile_dnsrow_Val);
    hipFree(d_Tile_dnsrow_idx);
    hipFree(d_Tile_dnscol_Val);
    hipFree(d_Tile_dnscol_idx);
    hipFree(d_coo_new_value);
    hipFree(d_coo_new_rowidx);
    hipFree(d_coo_new_matrix_ptr);
    hipFree(d_coo_new_matrix_colidx);

printf("--------------------------------!!-cuda spmv end-!!------------------------------------\n");

    Tile_destroy(matrixA_d);
    free(new_nnz_count_1);
    free(new_coo_colidx_1);
    free(new_coo_rowidx_1);
    free(new_coo_value_1);
    free(group_ptr_1);
    hipFree(d_new_nnz_count);
    hipFree(d_coo_new_colidx);
    hipFree(d_coo_new_value);
    hipFree(d_new_num);
    hipFree(d_new_coo_value);
    hipFree(d_new_coo_rowidx);


    return 0;
}
