#include "hip/hip_runtime.h"
#include "common.h"
#include "mmio_highlevel.h"
#include "utils.h"
#include "csr2tile.h"
#include "external/CSR5_cuda/anonymouslib_cuda.h"
#include "tilespmv_cpu.h"
#include "tilespmv_cuda.h"

# define INDEX_DATA_TYPE unsigned char


#define DEBUG_FORMATCOST 0


int main(int argc, char ** argv)
{

	if (argc < 2)
    {
        printf("Run the code by './test matrix.mtx'.\n");
        return 0;
    }
	
    printf("--------------------------------!!!!!!!!------------------------------------\n");

 	struct timeval t1, t2;
	int rowA;
	int colA;
	MAT_PTR_TYPE nnzA;
	int isSymmetricA;
    MAT_VAL_TYPE *csrValA;
    int *csrColIdxA;
    MAT_PTR_TYPE *csrRowPtrA;
	
    int device_id = 0;
    // "Usage: ``./spmv -d 0 mtx A.mtx'' for Ax=y on device 0"
    int argi = 1;

    // load device id
    char *devstr;
    if(argc > argi)
    {
        devstr = argv[argi];
        argi++;
    }

    if (strcmp(devstr, "-d") != 0) return 0;

    if(argc > argi)
    {
        device_id = atoi(argv[argi]);
        argi++;
    }
    printf("device_id = %i\n", device_id);


	char  *filename;
    filename = argv[3];
    printf("MAT: -------------- %s --------------\n", filename);

    // load mtx A data to the csr format
    gettimeofday(&t1, NULL);
    mmio_allinone(&rowA, &colA, &nnzA, &isSymmetricA, &csrRowPtrA, &csrColIdxA, &csrValA, filename);
    gettimeofday(&t2, NULL);
    double time_loadmat  = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("  input matrix A: ( %i, %i ) nnz = %i\n  loadfile time    = %4.5f sec\n", rowA, colA, nnzA, time_loadmat/1000.0);

	for (int i = 0; i < nnzA; i++)
	    csrValA[i] = i % 10;

    rowA = (rowA / BLOCK_SIZE) * BLOCK_SIZE;

    // set device
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    printf("---------------------------------------------------------------------------------------------\n");
    printf("Device [ %i ] %s @ %4.2f MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);



    Tile_matrix *matrixA = (Tile_matrix *)malloc(sizeof (Tile_matrix));

    //format conversion

    Tile_create(matrixA, 
                rowA, colA, nnzA,
                csrRowPtrA,
                csrColIdxA,
                csrValA);

	MAT_VAL_TYPE *x = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * colA);
	for (int i = 0; i < colA; i++)
	{
		x[i] = i % 10;
	}

    // compute reference results on a cpu core

	MAT_VAL_TYPE *y_golden = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
	for (int i = 0; i < rowA; i++)
	{
		MAT_VAL_TYPE sum = 0;
		for (int j = csrRowPtrA[i]; j < csrRowPtrA[i+1]; j++)
		{
			sum += csrValA[j] * x[csrColIdxA[j]];
		}
		y_golden[i] = sum;
	}




    //run CPU TileSpMV
    

    struct timeval cpu_tstart,cpu_tend;
    gettimeofday(&cpu_tstart, NULL);

	MAT_VAL_TYPE *y = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
    memset(y, 0, sizeof(MAT_VAL_TYPE) * rowA);

    int tilenum = matrixA->tilenum;


    int * ptroffset1 = (int *)malloc(sizeof(int) * tilenum);
    int * ptroffset2 = (int *)malloc(sizeof(int) * tilenum);
    memset(ptroffset1, 0, sizeof(int) * tilenum);
    memset(ptroffset2, 0, sizeof(int) * tilenum);

    int rowblkblock = 0;

    unsigned int * blkcoostylerowidx ;
    int * blkcoostylerowidx_colstart   ;
    int * blkcoostylerowidx_colstop ;
    int *multicoo_ptr = (int *)malloc((rowA + 1) * sizeof(int));

    int *multicoo_colidx ;
    MAT_VAL_TYPE *multicoo_val ;

    tilespmv_cpu(matrixA,
                ptroffset1,
                ptroffset2,
                &rowblkblock,
                &blkcoostylerowidx,
                &blkcoostylerowidx_colstart,
                &blkcoostylerowidx_colstop,
                rowA, colA, nnzA,
                csrRowPtrA,
                csrColIdxA,
                csrValA,
                x,
                y,
                y_golden
            );


  MAT_VAL_TYPE alpha = 1.0;
  memset(y, 0, sizeof(MAT_VAL_TYPE) * rowA);


//run GPU TilespMV

    call_tilespmv_cuda( filename,
                        matrixA,
                        ptroffset1,
                        ptroffset2,
                        rowblkblock,
                        blkcoostylerowidx,
                        blkcoostylerowidx_colstart,
                        blkcoostylerowidx_colstop,
                        rowA, colA, nnzA,
                        csrRowPtrA,
                        csrColIdxA,
                        csrValA,
                        alpha,
                        x,
                        y,
                        y_golden);



    //check results

    int error_count_cuda = 0;
    for (int i = 0; i < rowA; i++)
        if (abs(y_golden[i] - y[i]) > 0.01 * abs(y[i]))
        {
            error_count_cuda++;
            // cout<<"y_golden = "<<y_golden[i]<<" , "<<"y = "<<y[i]<<endl;
        }

    if (error_count_cuda == 0)
        cout << "Check... PASS!" << endl;
    else
        cout << "Check... NO PASS! error_count_cuda = "<< error_count_cuda<< endl;


    free(matrixA);
    free(csrValA);
    free(csrColIdxA);
    free(csrRowPtrA);

}
